#include "hip/hip_runtime.h"
/**********************************************************
 * @author  Pulkit Verma
 * @email   technopreneur[dot]pulkit[at]gmail[dot]com
 **********************************************************/
 
// The program takes two equal size vectors as input and outputs their vector sum

#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>

void CUDA_SAFE_CALL(hipError_t call)
{
    hipError_t ret = call;
    switch(ret)
    {
        case hipSuccess:
            break;
        default:
        {
            printf("%s in %s at line %d\n",hipGetErrorString(ret), __FILE__, __LINE__);
            exit(-1);
            break;
        }
    }
    return;
}

__global__ void vecAdd(float *in1, float *in2, float *out, int len) 
{
	int i = blockIdx.x * blockDim.x + threadIdx.x;

	if(i<len)
		out[i]=in1[i]+in2[i];

	return;
}

int main(int argc, char **argv) 
{
	int inputLength;
	float *hostInput1;
	float *hostInput2;
	float *hostOutput;
	float *deviceInput1;
	float *deviceInput2;
	float *deviceOutput;

	hostInput1 = ( float * )malloc(inputLength * sizeof(float));
	hostInput2 = ( float * )malloc(inputLength * sizeof(float));
	hostOutput = ( float * )malloc(inputLength * sizeof(float));

	scanf("%d ",&inputLength);

	for(int i=0;i<inputLength)
		scanf("%f ",&hostInput1[i]);

	for(int i=0;i<inputLength)
		scanf("%f ",&hostInput2[i]);	

	CUDA_SAFE_CALL( hipMalloc ( (void**) &deviceInput1, inputLength * sizeof(float)) );
	CUDA_SAFE_CALL( hipMalloc ( (void**) &deviceInput2, inputLength * sizeof(float)) );
	CUDA_SAFE_CALL( hipMalloc ( (void**) &deviceOutput, inputLength * sizeof(float)) );

	CUDA_SAFE_CALL( hipMemcpy ( deviceInput1, hostInput1, inputLength * sizeof(float), hipMemcpyHostToDevice) );  
	CUDA_SAFE_CALL( hipMemcpy ( deviceInput2, hostInput2, inputLength * sizeof(float), hipMemcpyHostToDevice) );  

	hipDeviceProp_t deviceProp;
	CUDA_SAFE_CALL( hipGetDeviceProperties(&deviceProp, 0) );
	dim3 DimGrid ( ceil(inputLength/(float)deviceProp.maxThreadsDim[0]), 1, 1);
	dim3 DimBlock ( deviceProp.maxThreadsDim[0], 1, 1);

	vecAdd <<< DimGrid, DimBlock >>> (deviceInput1, deviceInput2, deviceOutput, inputLength);
	hipDeviceSynchronize();
	CUDA_SAFE_CALL( hipMemcpy ( hostOutput, deviceOutput, inputLength * sizeof(float), hipMemcpyDeviceToHost) );  

	CUDA_SAFE_CALL( hipFree (deviceInput1) );
	CUDA_SAFE_CALL( hipFree (deviceInput2) );
	CUDA_SAFE_CALL( hipFree (deviceOutput) );

	for(int i=0;i<inputLength;i++)
		printf("%f ",hostOutput[i]);
	printf("\n");

	free(hostInput1);
	free(hostInput2);
	free(hostOutput);

	return 0;
}

